#include "hip/hip_runtime.h"
// Copyright (c) 2023 Forschungszentrum Jülich GmbH

#include <stdio.h>

// This is the function that runs on the GPU
__global__ void cuda_kernel(){

    // -- TODO (Ex2.2.e)
    // Let each CUDA thread to print its own ID in the form:
    // "Hello World from GPU! thread: X\n"
    printf("Hello World from GPU! thread: %d\n",threadIdx.x);

}


// The main program starts on the CPU
int main(int argc, char* argv[]) {

    // -- TODO (Ex2.2.e)
    // Print a "Hello World from the CPU\n"
    printf("Hello World from the CPU\n");

    // -- TODO (Ex2.2.e)
    // Launch 'cuda_kernel' on the GPU (with 1 block and 8 threads)
     hello<<<1, 8>>>();



    hipDeviceSynchronize(); 
    return 0;
}
